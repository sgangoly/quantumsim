/*This file is part of quantumsim. (https://github.com/brianzi/quantumsim)*/
/*(c) 2016 Brian Tarasinski*/
/*Distributed under the GNU GPLv3. See LICENSE.txt or https://www.gnu.org/licenses/gpl.txt*/

#include <hip/hip_runtime.h> 

//kernel to transform to pauli basis (up, x, y, down)
//to be run on a complete complex density matrix, once for each bit
//this operation is its own inverse (can also be used in opposite direction)
//run with a 2d grid of total size (2**no_qubits)^2
__global__ void bit_to_pauli_basis(double *complex_dm, unsigned int mask, unsigned int no_qubits) {
    const int x = (blockIdx.x *blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y *blockDim.y) + threadIdx.y;

    const double sqrt2 =  0.70710678118654752440;
    //const double sqrt2 =  1;

    if ((x >= (1 << no_qubits)) || (y >= (1 << no_qubits))) return;

    int b_addr = ((x|mask)<<no_qubits | (y&~mask)) << 1;
    int c_addr = ((x&~mask)<<no_qubits | (y|mask)) << 1;

    if (x&mask && (~y&mask)){
        double b = complex_dm[b_addr];
        double c = complex_dm[c_addr];
        complex_dm[b_addr] = (b+c)*sqrt2;
        complex_dm[c_addr] = (b-c)*sqrt2;
    }
    if ((~x&mask) && (y&mask)){
        b_addr+=1;
        c_addr+=1;
        double b = complex_dm[b_addr];
        double c = complex_dm[c_addr];
        complex_dm[b_addr] = (b+c)*sqrt2;
        complex_dm[c_addr] = (b-c)*sqrt2;
    }
}


//pauli_reshuffle
//this function collects the values from a complex density matrix in (0, x, iy, 1) basis
//and collects the real or values only; furthermore it rearranges the address bit order 
//from (d_state_bits, d_state_bits) to 
// (alpha_d, alpha_d-1, ..., alpha_0) where alpha = (00, 01, 10, 11) for 0, x, y, 1
//if direction = 0, the copy is performed from complex to real, otherwise from real to complex
__global__ void pauli_reshuffle(double *complex_dm, double *real_dm, unsigned int no_qubits, unsigned int direction) {

    const int x = (blockIdx.x *blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y *blockDim.y) + threadIdx.y;

    if ((x >= (1 << no_qubits)) || (y >= (1 << no_qubits))) return;


    //do we need imaginary part? That is the case if we have an odd number of bits for y in our adress (bit in y is 1, bit in x is 0)
    unsigned int v = ~x & y;


    unsigned int py = 0;
    while (v) {
        py += v&1;
        v >>= 1;
    }

    py = py & 0x3;

    //short version: while (v>1) { v = (v >> 1) ^ v ;}
    //bit bang version
    /*v ^= v >> 1;*/
    /*v ^= v >> 2;*/
    /*v = (v & 0x11111111U) * 0x11111111U;*/
    /*v = (v >> 28) & 1;*/

    const unsigned int addr_complex = (((x << no_qubits) | y) << 1) + (py&1);


    //the adress in pauli basis is obtained by interleaving
    unsigned int addr_real = 0;
    for (int i = 0; i < 16; i++) { 
          addr_real |= (x & 1U << i) << i | (y & 1U << i) << (i + 1);
    }
    

    if(direction == 0) {
        real_dm[addr_real] = ((py==3 || py==2)? -1 : 1)*complex_dm[addr_complex];
    }
    else {
        complex_dm[addr_complex] = ((py==3 || py == 2)? -1 : 1)*real_dm[addr_real];
    }
}



//Apply a general pauli transfer matrix, to (up to) two subsystems (arbitrary dimension)
//a is the msb, b is the lsb.
//If the PTM is diagonal, this works in-place, i.e. with dm_in == dm_out. Otherwise NOT!
//You need to give the dimensions of two of the intermediate bystander spaces. 

//it is also important that a must be the msb in the ptm, but also the msb in the density matrix.
//if not, you must reshape the ptm (switch a and b) before calling the kernel.
__global__ void two_qubit_general_ptm(
        double *dm_in, double *dm_out,
        double *ptm_g, 
        unsigned int dim_a_in, 
        unsigned int dim_b_in, 
        unsigned int dim_z, unsigned int dim_y,
        unsigned int dim_rho) {

    /*const unsigned int idx = threadIdx.x + blockIdx.x*blockDim.x;*/
    /*if (idx >= dim_rho) return;*/


    //(structure worked out in Notebook II, p. 203 ff)
        
    //blockDim.x = dim_a_out
    //blockDim.y = dim_b_out
    //blockDim.z = d_internal (arbitrary, to make blocksize and shared memory reasonable)

    const unsigned int dim_a_out = blockDim.x;
    const unsigned int dim_b_out = blockDim.y;
    const unsigned int d_internal = blockDim.z;

    const unsigned int ax = threadIdx.x;
    const unsigned int bx = threadIdx.y;
    const unsigned int zx = threadIdx.z;
    const unsigned int ix = blockIdx.x;

    unsigned int xyz, x, y, z;
    xyz = ix*d_internal + zx;
    z = xyz % dim_z;
    xyz /= dim_z;
    y = xyz % dim_y;
    xyz /= dim_y;
    x = xyz;
    // can do/need termination statement here?

    // external memory required: (blockDim.x + dim_a*dim_b**2) double floats
    extern __shared__ double ptm[];
    double *data = &ptm[dim_a_in*dim_b_in*dim_a_out*dim_b_out]; 

    // load ptm to shared memory 

    const int row = (ax*dim_b_out + bx)*dim_b_in*dim_a_in;
    for(int g = zx; g < dim_a_in*dim_b_in; g += d_internal) {
        ptm[row + g] = ptm_g[row + g];
    }

    // load data to memory
    const int column = zx*dim_a_in*dim_b_in;
    unsigned int addr_in;
    for(int ai = ax; ai < dim_a_in; ai += dim_a_out) {
        for(int bi = bx; bi < dim_b_in; bi += dim_b_out) {
            addr_in = (((x*dim_a_in + ai)*dim_y + y)*dim_b_in + bi)*dim_z + z;
            data[column + ai*dim_b_in + bi] = dm_in[addr_in];
        }
    }

    //done loading
    __syncthreads();

    //calculate the vector product
    double acc=0.0;
    for(int delta=0; delta < dim_a_in*dim_b_in; delta++) {
        acc += ptm[row + delta]*data[column + delta];
    }

    //upload back to global memory
    __syncthreads();
    const int addr_out = (((x*dim_a_out + ax)*dim_y + y)*dim_b_out + bx)*dim_z + z;
    dm_out[addr_out] = acc;
}


// apply a 4x4 pauli transfer matrix (in 0, x, y, 1 basis)
// to the specified qubit
__global__ void single_qubit_ptm(double *dm, double *ptm_g,  unsigned int bit, unsigned int no_qubits) {
    const unsigned int x = threadIdx.x;
    const unsigned int high_x = blockIdx.x * blockDim.x;

    if (high_x + x >= (1 << (2*no_qubits))) return;

    //the two lowest bits of thread id are used to index the target bit,
                                                //      xx <- target bit
    int high_mask = ~ ( (1 << (2*bit+2)) - 1 ); // 1111100000000
    int low_mask  = ~high_mask & (~0x3);        // 0000011111100

    int pos = high_x | x;
    int global_from = (pos & high_mask) | ((pos & 0x3) << (2*bit)) | ((pos & low_mask)>>2);

    extern __shared__ double ptm[];
    double *data = &ptm[16]; //need blockDim.x double floats

    //first fetch the transfer matrix to shared memory
    if(x < 16) ptm[x] = ptm_g[x];

    if(no_qubits < 2) { //what a boring situation
        ptm[x+4] = ptm_g[x+4];
        ptm[x+8] = ptm_g[x+8];
        ptm[x+12] = ptm_g[x+12];
    }

    //fetch block to shared memory
    data[x] = dm[global_from];
    __syncthreads();

    //do calculation

    int row = x & 0x3;
    int idx = x & ~0x3;

    double acc = 0;

    acc += ptm[4*row    ] * data[idx    ];
    acc += ptm[4*row + 1] * data[idx + 1];
    acc += ptm[4*row + 2] * data[idx + 2];
    acc += ptm[4*row + 3] * data[idx + 3];

    //upload back to global memory
    __syncthreads();
    dm[global_from] = acc;
}


//apply a two-qubit (16x16) ptm to two qubits. This can be done in-place and with fast memory calculation.
__global__ void two_qubit_ptm(double *dm, double *ptm_g, unsigned int bit0, unsigned int bit1, unsigned int no_qubits) {
    const unsigned int x = threadIdx.x;
    const unsigned int high_x = blockIdx.x * blockDim.x;

    extern __shared__ double ptm[];
    double *data = &ptm[256]; //need blockDim.x double floats

    // the lowest to bits of x are used to address bit0, the next two are used to address bit1 
    // global address = <- pos = 
    // aaaxxbbbbyycccc  <- aaabbbbccccxxyy

    int higher_bit = max(bit0, bit1);
    int lower_bit = min(bit0, bit1);
    int high_mask = ~ ( (1 << (2*higher_bit+2)) - 1 ); //a mask (of pos)
    int mid_mask = (~ ( (1 << (2*lower_bit + 4)) - 1)) & (~high_mask);  //b mask
    int low_mask  = ~(high_mask | mid_mask) & (~0xf);  //c mask

    int pos = high_x | x;
    int global_from = 
              (pos & high_mask) 
            | ((pos & mid_mask) >> 2)
            | ((pos & low_mask) >> 4)
            | ((pos & 0x3) << (2 * bit0))  
            | (((pos & 0xc) >>2)  << (2 * bit1));

    //fetch ptm to shared memmory
    //need to fetch several values per thread if blockDim.x is less than 256 (only for small dms...)
    for(int i=0; i < 256; i+=blockDim.x) {
        if(i+x < 256) {
            ptm[i+x] = ptm_g[i+x];
        }
    }
    if (high_x + x >= (1 << (2*no_qubits))) return;


    //fetch data block to shared memory
    data[x] = dm[global_from];
    __syncthreads();

    unsigned int row = x & 0xf;
    unsigned int idx = x & ~0xf;

    double acc=0;
    for(int i=0; i<16; i++) {
        acc += ptm[16*row + i]*data[idx+i];
    }


    __syncthreads();
    dm[global_from] = acc;

}


//copy the two diagonal blocks of one ancilla into reduced density matrices
//the qubit index is passed as an integer, not as a bitmask!
__global__ void dm_reduce(double *dm, unsigned int bit, double *dm0, unsigned int state,
        unsigned int no_qubits) {

    const int addr = blockIdx.x*blockDim.x + threadIdx.x;

    if(addr >= (1<< (2*no_qubits))) return;

    const int low_mask = (1 << (2*bit))-1;      //0000011111
    const int high_mask = (~low_mask) << 2;     //1110000000

    if(((addr >> (2*bit)) & 0x3) == state*0x3) {
        dm0[ (addr & low_mask) | ((addr & high_mask) >> 2) ] = dm[addr];
    }
}


//multitake
//given a list of index lists `idx` in sparse format
//idx_j = flatten(idx), idx_i = cumsum(len(idx))
//and in and out array with shapes
//as well as dim = len(inshape) = len(outshape)
//set out = in[np.ix_(idx)]
__global__ void multitake(double *in, double *out, 
        unsigned int *idx_i, unsigned int *idx_j, 
        unsigned int *inshape, unsigned int *outshape, 
        unsigned int dim) {

    unsigned int addr_out, addr_in, s;
    unsigned int i, ia, ja;

    int acc;

    acc = addr_out = blockDim.x*blockIdx.x + threadIdx.x;
    addr_in = 0;
    s = 1;

    for(i=dim; i > 0;) {
        i--;
        /*ia = rint(remquo((double)acc, (double)outshape[i], &acc));*/
        ia = acc % outshape[i];
        acc = acc / outshape[i];
        ja = idx_j[idx_i[i] + ia];
        addr_in += ja*s;
        s *= inshape[i];
    }

    // guard 
    if(acc == 0) 
        out[addr_out] = in[addr_in];
}



//get_diagonal kernel
//copy the diagonal elements to out, in order to do effective 
//calculation of subtraces.
//run over a 1x9 grid!
__global__ void get_diag(double *dm9, double *out, unsigned int no_qubits) {
    int x = (blockIdx.x *blockDim.x) + threadIdx.x;

    if (x >= (1 << no_qubits)) return;
    unsigned int addr_real = 0;
    for (int i = 0; i < 16; i++) { 
          addr_real |= (x & 1U << i) << i | (x & 1U << i) << (i + 1);
    }
    out[x] = dm9[addr_real];
}

//trace kernel. Calculate the sum of a diagonal, must run in one block!
//shared memory: 2**no_qubits doubles
//if bit is positive or zero, diag[0] and diag[1] will hold the partial traces of this bit being one/zero (!note the switch)
//if bit is -1, diag[0] will hold the full trace.
__global__ void trace(double *diag, int bit) { 
    unsigned int x = threadIdx.x;
    unsigned int mask = 0;

    if(bit >= 0) {
        mask = 1 << bit;
    }

    extern __shared__ double s_diag[];
    s_diag[x] = diag[x];
    __syncthreads(); 

    double a;

    for(unsigned int i=1; i < blockDim.x; i <<= 1) {
        if(i != mask && i <= x) { 
            a = s_diag[x-i];
        
        }
        __syncthreads();
        if(i != mask && i <= x) { 
            s_diag[x] += a;
        }
        __syncthreads();
    }

    __syncthreads();
    //copy result back
    if(x == 0) {
        diag[blockIdx.x] = s_diag[blockDim.x - 1];
        return;
    }
    if(x == 1 && bit >= 0) {
        diag[blockIdx.x + 1] = s_diag[blockDim.x - 1 - mask];
        return;
    }
}

//swap kernel
//exchange two qubits. The only purpose of this kernel is to arrange a certain qubit as to be the most significant so that
//projection is trivial. Actual swap gates should be implemented by a two-qubit ptm, (or by relabeling if perfect)!
__global__ void swap(double *dm, unsigned int bit1, unsigned int bit2, unsigned int no_qubits) {
    unsigned int addr = threadIdx.x + blockDim.x*blockIdx.x;

    if (addr >= (1<<2*no_qubits)) return;

    unsigned int bit1_mask = (0x3 << (2*bit1));
    unsigned int bit2_mask = (0x3 << (2*bit2));
    
    unsigned int addr2 = ( addr & ~(bit1_mask | bit2_mask)) |
        ((addr & bit1_mask) << (2*(bit2 - bit1))) |
        ((addr & bit2_mask) >> (2*(bit2 - bit1)));
   
    double t;
    if (addr > addr2) {
        t = dm[addr2];
        dm[addr2] = dm[addr];
        dm[addr] = t;
    }
}
